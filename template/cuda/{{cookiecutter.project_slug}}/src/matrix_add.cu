#include "hip/hip_runtime.h"
#include "cuda_utils.h"

__global__ void matrixAdd(const float* A, const float* B, float* C, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols) {
        int idx = row * cols + col;
        C[idx] = A[idx] + B[idx];
    }
}

void matrixAddHost(const float* A, const float* B, float* C, int rows, int cols) {
    size_t size = rows * cols * sizeof(float);

    float* d_A;
    float* d_B;
    float* d_C;

    CUDA_CHECK(hipMalloc(&d_A, size));
    CUDA_CHECK(hipMalloc(&d_B, size));
    CUDA_CHECK(hipMalloc(&d_C, size));

    CUDA_CHECK(hipMemcpy(d_A, A, size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, B, size, hipMemcpyHostToDevice));

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((cols + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (rows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matrixAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, rows, cols);
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));
}
