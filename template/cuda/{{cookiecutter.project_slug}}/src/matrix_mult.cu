#include "cuda_utils.h"

void matrixMultHost(const float* A, const float* B, float* C, int rowsA, int colsA, int colsB) {
    size_t sizeA = rowsA * colsA * sizeof(float);
    size_t sizeB = colsA * colsB * sizeof(float);
    size_t sizeC = rowsA * colsB * sizeof(float);

    float* d_A;
    float* d_B;
    float* d_C;

    CUDA_CHECK(hipMalloc(&d_A, sizeA));
    CUDA_CHECK(hipMalloc(&d_B, sizeB));
    CUDA_CHECK(hipMalloc(&d_C, sizeC));

    CUDA_CHECK(hipMemcpy(d_A, A, sizeA, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, B, sizeB, hipMemcpyHostToDevice));

    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    const float alpha = 1.0f;
    const float beta = 0.0f;
    CUBLAS_CHECK(hipblasSgemm(handle,
                            HIPBLAS_OP_N, HIPBLAS_OP_N,
                            colsB, rowsA, colsA,
                            &alpha,
                            d_B, colsB,
                            d_A, colsA,
                            &beta,
                            d_C, colsB));

    CUDA_CHECK(hipMemcpy(C, d_C, sizeC, hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));

    CUBLAS_CHECK(hipblasDestroy(handle));
}
