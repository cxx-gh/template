#include "cuda_utils.h"

// Function to perform matrix multiplication on GPU using cuBLAS
void multiplyMatricesOnGPU(const float* hostMatrixA, const float* hostMatrixB, float* hostResultMatrix,
                           int numRowsA, int numColsA, int numColsB) {
    // Calculate sizes in bytes for each matrix
    size_t byteSizeA = numRowsA * numColsA * sizeof(float);
    size_t byteSizeB = numColsA * numColsB * sizeof(float);
    size_t byteSizeC = numRowsA * numColsB * sizeof(float);

    // Declare pointers for device (GPU) memory
    float *deviceMatrixA, *deviceMatrixB, *deviceResultMatrix;

    // Allocate memory on the GPU
    CUDA_CHECK(hipMalloc(&deviceMatrixA, byteSizeA));
    CUDA_CHECK(hipMalloc(&deviceMatrixB, byteSizeB));
    CUDA_CHECK(hipMalloc(&deviceResultMatrix, byteSizeC));

    // Copy input matrices from host to device
    CUDA_CHECK(hipMemcpy(deviceMatrixA, hostMatrixA, byteSizeA, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(deviceMatrixB, hostMatrixB, byteSizeB, hipMemcpyHostToDevice));

    // Create cuBLAS handle
    hipblasHandle_t cublasHandle;
    CUBLAS_CHECK(hipblasCreate(&cublasHandle));

    // Set up parameters for hipblasSgemm
    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Perform matrix multiplication using cuBLAS
    CUBLAS_CHECK(hipblasSgemm(cublasHandle,
                             HIPBLAS_OP_N, HIPBLAS_OP_N,
                             numColsB, numRowsA, numColsA,
                             &alpha,
                             deviceMatrixB, numColsB,
                             deviceMatrixA, numColsA,
                             &beta,
                             deviceResultMatrix, numColsB));

    // Copy the result back to host memory
    CUDA_CHECK(hipMemcpy(hostResultMatrix, deviceResultMatrix, byteSizeC, hipMemcpyDeviceToHost));

    // Clean up: Free GPU memory and destroy cuBLAS handle
    CUDA_CHECK(hipFree(deviceMatrixA));
    CUDA_CHECK(hipFree(deviceMatrixB));
    CUDA_CHECK(hipFree(deviceResultMatrix));
    CUBLAS_CHECK(hipblasDestroy(cublasHandle));
}
